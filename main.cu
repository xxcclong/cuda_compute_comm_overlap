/* Includes, system */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <memory>
#include <vector>
#include <thread>
#include <assert.h>
#include <functional>

/* Includes, cuda */
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <nccl.h>

// 1: element wise sqrt 2: element wise mul others: cublas gemm
#define TEST_ALGO 2
#define WAIT_EVENT

/* Matrix size */
#define N (32 * 4)
#define M (32 * 4)

dim3 cuda_threads(64);
dim3 cuda_grid(N / 64);

#define GPUS (4)
#define ITERATIONS (200)
#define PRERUN_ITER (100)
#define COMPUTE_TIME 20
#define COMM_TIME 1
//float *h_C_ref;
float* d_A[GPUS];
float* d_B[GPUS];
float* d_C[GPUS];
float* d_D[GPUS];
float alpha = 1.0f;
float beta = 0.0f;
int n2 = N * N;
int i;

enum NCCL_MODE {
    ASYNC = 0,
    SYNC = 1,
    ONE_STREAM = 2,
    NO_COMM = 3,
    NO_COMPUTE = 4
};


std::unique_ptr<ncclComm_t[]> comms = nullptr;
std::unique_ptr<hipStream_t[]> nccl_streams = nullptr;
std::unique_ptr<hipStream_t[]> blas_streams = nullptr;
size_t timestamp() {
    using namespace std::chrono;
    return duration_cast<microseconds>(
               high_resolution_clock::now().time_since_epoch()).count();
}

void init_nccl() {
    comms.reset(new ncclComm_t[GPUS]);
    nccl_streams.reset(new hipStream_t[GPUS]);
    blas_streams.reset(new hipStream_t[GPUS]);
    ncclUniqueId nccl_id;
    ncclGetUniqueId(&nccl_id);
    ncclGroupStart();
    for (size_t i = 0; i < GPUS; ++i) {
        hipSetDevice(i);
        hipStreamCreate(nccl_streams.get() + i);
        ncclCommInitRank(comms.get() + i, GPUS, nccl_id, i);
        hipStreamCreate(blas_streams.get() + i);
    }
    ncclGroupEnd();
}

int init_data(int dev) {
    float* ha;
    float* hb;
    float* hc;
    float* hd;
    //float *h_C_ref;
    d_A[dev] = 0;
    d_B[dev] = 0;
    d_C[dev] = 0;
    d_D[dev] = 0;
    //float *da = *d_A[dev] = 0;
    //float *db = *d_B[dev] = 0;
    //float *dc = *d_C[dev] = 0;
    hipSetDevice(dev);
    /* Allocate host memory for the matrices */
    ha = reinterpret_cast<float*>(malloc(n2 * sizeof(ha[0])));
    hb = reinterpret_cast<float*>(malloc(n2 * sizeof(hb[0])));
    hc = reinterpret_cast<float*>(malloc(n2 * sizeof(hc[0])));
    hd = reinterpret_cast<float*>(malloc(M * sizeof(hd[0])));

    /* Fill the matrices with test data */
    for (i = 0; i < n2; i++) {
        ha[i] = rand() / static_cast<float>(RAND_MAX);
        hb[i] = rand() / static_cast<float>(RAND_MAX);
        hc[i] = rand() / static_cast<float>(RAND_MAX);
    }
    for (i = 0; i < M; i++)
        hd[i] = rand() / static_cast<float>(RAND_MAX);


    /* Allocate device memory for the matrices */
    if (hipMalloc(reinterpret_cast<void**>(&d_A[dev]), n2 * sizeof(d_A[dev][0])) !=
            hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_B[dev]), n2 * sizeof(d_B[dev][0])) !=
            hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate B)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_C[dev]), n2 * sizeof(d_C[dev][0])) !=
            hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate C)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc(reinterpret_cast<void**>(&d_D[dev]), M * sizeof(d_D[dev][0])) !=
            hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate D)\n");
        return EXIT_FAILURE;
    }


    /* Initialize the device matrices with the host matrices */
    hipblasSetVector(n2, sizeof(ha[0]), ha, 1, d_A[dev], 1);
    hipblasSetVector(n2, sizeof(hb[0]), hb, 1, d_B[dev], 1);
    hipblasSetVector(n2, sizeof(hc[0]), hc, 1, d_C[dev], 1);
    return 0;
}

int destroy_data(int dev) {
    //float *h_C_ref;
    float* da = d_A[dev];
    float* db = d_B[dev];
    float* dc = d_C[dev];
    float* dd = d_D[dev];
    /* Memory clean up */

    if (hipFree(da) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (A)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(db) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (B)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(dc) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (C)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(dd) != hipSuccess) {
        fprintf(stderr, "!!!! memory free error (C)\n");
        return EXIT_FAILURE;
    }
    return 0;
}

__global__ void eleSqrt(float *a, float* c)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] += sqrtf(fabsf(a[idx]));
}

__global__ void eleMul(float *a, float* b, float* c)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] += a[idx] * b[idx];
}


inline void print_compute_func()
{
#ifndef TEST_ALGO
    fprintf(stderr, "algo: cublasSgemm\n");
#elif TEST_ALGO == 1 
    fprintf(stderr, "algo: Sqrt\n");
#elif TEST_ALGO == 2 
    fprintf(stderr, "algo: Element Wise Mul\n");
#else
    fprintf(stderr, "algo: cublasSgemm\n");
#endif
}


inline void compute_func(int dev, hipblasHandle_t* handle, hipStream_t* s)
{
#ifndef TEST_ALGO
    cublasSgemm(*handle, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha, d_A[dev],
                             N, d_B[dev], N, &beta, d_C[dev], N);
#elif TEST_ALGO == 1 
    eleSqrt<<<cuda_grid, cuda_threads, 0, *s>>>(d_A[dev], d_C[dev]);
#elif TEST_ALGO == 2 
    eleMul<<<cuda_grid, cuda_threads, 0, *s>>>(d_A[dev], d_B[dev], d_C[dev]);
#else
    cublasSgemm(*handle, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha, d_A[dev],
                             N, d_B[dev], N, &beta, d_C[dev], N);
#endif
}

int prerun(int dev = 0) {
    hipblasStatus_t status;
    hipblasHandle_t handle;

    auto& blas_stream = *(blas_streams.get() + dev);
    hipSetDevice(dev);

    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    hipblasSetStream(handle, blas_stream);

    /* Performs operation using cublas */
    auto& nccl_stream = *(nccl_streams.get() + dev);
    hipEvent_t start_event, stop_event;
    float compute_time = 0, comm_time = 0, temp_time = 0;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    // run computation
    for (int i = 0; i < PRERUN_ITER; ++i) {
        hipEventRecord(start_event, 0);
        compute_func(dev, &handle, &blas_stream);
        hipEventRecord(stop_event, 0);
        hipEventSynchronize(stop_event);
        hipEventElapsedTime(&temp_time, start_event, stop_event);
        compute_time += temp_time;
    }

    // run comm
    for (int i = 0; i < PRERUN_ITER; ++i) {
        hipEventRecord(start_event, 0);
        ncclAllReduce(d_D[dev], d_D[dev], M, ncclFloat, ncclSum, *(comms.get() + dev), nccl_stream);
        hipEventRecord(stop_event, 0);
        hipEventSynchronize(stop_event);
        hipEventElapsedTime(&temp_time, start_event, stop_event);
        comm_time += temp_time;
    }
    if (dev == 0)
        fprintf(stderr, "compute kernel time %fms\ncomm kernel time %fms\nin theory\nall compute time %fms\nall comm time %fms\ncompute / comm %f\n\n",
                compute_time / PRERUN_ITER,
                comm_time / PRERUN_ITER,
                compute_time / PRERUN_ITER * COMPUTE_TIME * ITERATIONS,
                comm_time / PRERUN_ITER * COMM_TIME * ITERATIONS,
                compute_time * COMPUTE_TIME / (comm_time * COMM_TIME));

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
    return 0;
}


/* Main */
int worker(int dev, int nccl_mode) {
    hipblasStatus_t status;

    hipblasHandle_t handle;
    auto& blas_stream = *(blas_streams.get() + dev);
    hipSetDevice(dev);

    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    hipblasSetStream(handle, blas_stream);

    /* Performs operation using cublas */
    auto& nccl_stream = *(nccl_streams.get() + dev);
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    size_t start = timestamp();
    hipEventRecord(start_event, 0);
    if (nccl_mode == NCCL_MODE::ONE_STREAM) {
        for (size_t i = 0; i < ITERATIONS; ++i) {
            for (int temp = 0; temp < COMPUTE_TIME; ++temp)
                compute_func(dev, &handle, &blas_stream);
            for (int temp = 0; temp < COMM_TIME; ++temp)
                ncclAllReduce(d_D[dev], d_D[dev], M, ncclFloat, ncclSum, *(comms.get() + dev), blas_stream);
        }
        //cudaStreamSynchronize(blas_stream);
    } else {
        // nccl_mode is ASYNC_NCCL or SYNC_NCCL
        for (size_t i = 0; i < ITERATIONS; ++i) {
            if (nccl_mode != NO_COMPUTE) {
                for (int temp = 0; temp < COMPUTE_TIME; ++temp)
                    compute_func(dev, &handle, &blas_stream);
            }



            if (nccl_mode != NO_COMM) {
                for (int temp = 0; temp < COMM_TIME; ++temp) {
                    ncclAllReduce(d_D[dev], d_D[dev], M, ncclFloat, ncclSum, *(comms.get() + dev), nccl_stream);
                    if (nccl_mode == SYNC) {
                        hipStreamSynchronize(nccl_stream);
                    }
                }
            } else {
                for (int temp = 0; temp < COMM_TIME; ++temp) {
                    ncclAllReduce(d_D[dev], d_D[dev], 1, ncclFloat, ncclSum, *(comms.get() + dev), nccl_stream);
                    if (nccl_mode == SYNC) {
                        hipStreamSynchronize(nccl_stream);
                    }
                }
            }
        }
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    float event_overall_time = 0;
    hipEventElapsedTime(&event_overall_time, start_event, stop_event);
    fprintf(stderr, "device: [%d], %d iterations spent: cputime [%.2f ms] eventtime [%.2f ms] \n", dev, ITERATIONS, (timestamp() - start) / 1000.0, event_overall_time);
    //fprintf(stderr, "device: [%d], %d iterations spent: [%d ms]\n", dev, ITERATIONS, (timestamp()-start)/1000);


    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
    return 0;
}

int worker_with_wait(int dev, int nccl_mode) {
    hipblasStatus_t status;

    hipblasHandle_t handle;
    auto& blas_stream = *(blas_streams.get() + dev);
    hipSetDevice(dev);

    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    hipblasSetStream(handle, blas_stream);

    /* Performs operation using cublas */
    auto& nccl_stream = *(nccl_streams.get() + dev);
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    std::vector<hipEvent_t> compute_events;
    compute_events.reserve(ITERATIONS);
    for(int i = 0; i < ITERATIONS; ++i) {
        hipEventCreateWithFlags(&compute_events[i], hipEventDisableTiming);
    }
    size_t start = timestamp();
    hipEventRecord(start_event, 0);

    for (size_t i = 0; i < ITERATIONS; ++i) {
        for (int temp = 0; temp < COMPUTE_TIME; ++temp){
            compute_func(dev, &handle, &blas_stream);
        }
        hipEventRecord(compute_events[i], blas_stream);
        if(i > 0) hipStreamWaitEvent(nccl_stream, compute_events[i - 1], 0);
        for (int temp = 0; temp < COMM_TIME; ++temp) {
            ncclAllReduce(d_D[dev], d_D[dev], M, ncclFloat, ncclSum, *(comms.get() + dev), nccl_stream);
        }
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    float event_overall_time = 0;
    hipEventElapsedTime(&event_overall_time, start_event, stop_event);
    fprintf(stderr, "[wait mode] device: [%d], %d iterations spent: cputime [%.2f ms] eventtime [%.2f ms] \n", dev, ITERATIONS, (timestamp() - start) / 1000.0, event_overall_time);


    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return EXIT_FAILURE;
    }
    return 0;
}

int main(int argc, char** argv) {
    if (argc != 2) {
        printf("USAGE: ./a.out 2 # 0 sync, 1 async, 2 one stream, 3 only compute, 4 only comm, 5 wait mode\n");
        return -1;
    }

    init_nccl();
    for (int i = 0; i < GPUS; ++i) {
        init_data(i);
    }
    std::vector<std::thread> threads;
    std::vector<std::thread> prerun_threads;
    int nccl_mode = atoi(argv[1]);
    printf("nccl mode %d\n", nccl_mode);
    print_compute_func();
    for (int i = 0; i < GPUS; ++i) {
        std::thread t(std::bind(&prerun, i));
        prerun_threads.push_back(std::move(t));
    }
    for (auto& t : prerun_threads) {
        t.join();
    }
    size_t start = timestamp();
    if(nccl_mode != 5){
        for (int i = 0; i < GPUS; ++i) {
            std::thread t(std::bind(&worker, i, nccl_mode));
            threads.push_back(std::move(t));
        }
        for (auto& t : threads) {
            t.join();
        }
    }
    else {
        for (int i = 0; i < GPUS; ++i) {
            std::thread t(std::bind(&worker_with_wait, i, nccl_mode));
            threads.push_back(std::move(t));
        }
        for (auto& t : threads) {
            t.join();
        }   
    }
    fprintf(stderr, "nccl mode: [%d], spent: [%.2f ms]\n", nccl_mode, (timestamp() - start) / 1000.0);


    for (int i = 0; i < GPUS; ++i) {
        destroy_data(i);
    }
    return 0;
}
